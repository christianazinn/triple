#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/partition.h>

using namespace std;

// 2^64-1 - if you want to fix the overflow issues you'll have to somehow cap values that overflow this.
#define absmax 18446744073709551615

// This is generally a mouthful to write.
#define pr thrust::pair<uint32_t, uint32_t>


/* ------------------------------------------------------------------------------------------------------------------
 * 
 * TRIPLE.CU version 1.0 written 2024-01-20 =========================================================================
 * 
 * This is a program to find triples of k-gonal numbers that have the property D(n).
 * Authored by Christian Zhou-Zheng, with help from Sounak Bagchi.
 * Created for number theory research at Euler Circle under Simon Rubenstein-Salzedo.
 * 
 * USER GUIDE -------------------------------------------------------------------------------------------------------
 * 
 * Everything that should be altered during use is in the "PARAMETERS" section below.
 * Change the parameters k and n accordingly.
 * Change the parameters abound, bbound, and cbound to change the search space.
 *   - Practical bounds on each (you run out of memory) are 10^5 for abound and bbound, and 10^10 for cbound.
 * You can also change the number of blocks and threads per block for your GPU.
 * When running this from the command line, on Linux (which you have to be on to run this as far as I'm aware), you
 *   can put "time" in front of it to benchmark runtime.
 * 
 * IMPLEMENTATION NOTES ---------------------------------------------------------------------------------------------
 * 
 * The program is split into two parts: checkPair and findc. checkPair takes an a and a b and checks if they satisfy
 *   D(n). If they do, it outputs the pair (a,b) to a thrust::device_vector. This section is known to work.
 * findc takes the output of checkPair and checks if the pair (a,b) satisfies D(n) with a c within cbound. If it does,
 *   it outputs c to a thrust::device_vector. This section is known to have issues.
 * There's a bit of intermediate processing in main() because due to how CUDA multithreading works, you can't have a
 *   dynamically resized output like a vector (even if thrust::device_vector masquerades as one, you can only modify)
 *   it from host code). So I have a full array of size abound*bbound (which is what causes the memory overflow that
 *   locks abound and bbound at 10^5) which has one position for each potential pair, each of which only gets written
 *   to if necessary. This results in a lot of waste that I can't figure out how to reduce. I do use 
 *   thrust::stable_partition to remove all the pairs with zeros, creating a smaller thrust::device_vector that I can 
 *   pass to findc, but I don't know how to reduce the size in checkPair.
 * 
 * KNOWN BUGS/NOTES -------------------------------------------------------------------------------------------------
 * 
 * I haven't actually tested the program for n != 1. It should work regardless.
 * Very large values of c/cbound will cause overflow issues with the eventual output. This makes no sense to me
 *   but it is what it is, and I think the <= in the final "print to stdout" section in main() fixes it.
 * Again, 10^5 is the practical limit for abound and bbound, due to the reasons detailed above. This can be fixed.
 * 
 * TODOS ------------------------------------------------------------------------------------------------------------
 * 
 * Ensure the program works for n != 1.
 * Fix the overflow issues with values of c.
 * Allow for a larger search space on abound and bbound by reducing the size of the array written to. Look into
 *   stream compaction. From Lemon#3040 on Discord:
 *   "Youll have to implement your own resizable vector
 *    The two choices is to either allocate what you are certain to be enough, then push the pairs there (if you dont
 *      care about ordering then you can simply increment an atomic, it wont hurt performance(at least if you 
 *      coalesce them by doing warp vote count, then have leader do the atomic)), and if it wasnt enough, you will 
 *      have to reallocate and run it again
 *    The problem is that you cant really do allocations from gpu
 *    You can preallocate a large buffer and then write a custom allocator, but it has its own problems
 *    So yea, resizable vectors are a lie
 *    Just get the upper bounds on output size, then use an atomic increment" 
 *   In particular, see https://stackoverflow.com/questions/34059753/cuda-stream-compaction-algorithm.
 * 
 * ------------------------------------------------------------------------------------------------------------------
 * Below here, change your parameters.
// PARAMETERS ---------------------------------------------------------------------------------------------------- */
#define k 7             // DEFAULT k=3 5 7 9
#define n 9             // DEFAULT n=1 or 9 for heptagonals
#define abound 1000     // DEFAULT 1000
#define bbound 1000     // DEFAULT 1000
#define cbound 10000000 // DEFAULT 10000000
#define boxsize 200     // DEFAULT 200 LIMIT 1024
#define threadsize 1000 // DEFAULT 1000 LIMIT 1024


// SQUARE-DETERMINING -----------------------------------------------------------------------------------------------
// Thanks to Norbert Juffa for this section, https://forums.developer.nvidia.com/t/integer-square-root/198642.
// This is imported code to quickly determine the floored integer square root of an integer.
/*
  Copyright (c) 2021, Norbert Juffa
  All rights reserved.

  Redistribution and use in source and binary forms, with or without 
  modification, are permitted provided that the following conditions
  are met:

  1. Redistributions of source code must retain the above copyright 
     notice, this list of conditions and the following disclaimer.

  2. Redistributions in binary form must reproduce the above copyright
     notice, this list of conditions and the following disclaimer in the
     documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
  A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
  HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
  SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
  LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
  DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
  THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT 
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
// Fast multiplication of two unsigned 32-bit integers with 64-bit result.
__device__ unsigned long long int umul_wide (unsigned int a, unsigned int b) {
    unsigned long long int r;
    asm ("mul.wide.u32 %0,%1,%2;\n\t" : "=l"(r) : "r"(a), "r"(b));
    return r;
}

// Compute the integer square root of a 64-bit unsigned integer by multiplying it with its inverse square root
// (which can be computed much faster), then applying one Newton-Raphson iteration and flooring.
__device__ uint64_t isqrtll (uint64_t a) {
    uint64_t rem, arg;
    uint32_t b, r, s, scal;

    arg = a;
    // Normalize argument
    scal = __clzll (a) & ~1;
    a = a << scal;
    b = a >> 32;
    // Approximate rsqrt accurately. Make sure it's an underestimate!
    float fb, fr;
    fb = (float)b;
    asm ("rsqrt.approx.ftz.f32 %0,%1; \n\t" : "=f"(fr) : "f"(fb));
    r = (uint32_t) fmaf (1.407374884e14f, fr, -438.0f);
    // Compute sqrt(a) as a * rsqrt(a).
    s = __umulhi (r, b);
    // NR iteration combined with back multiply.
    s = s * 2;
    rem = a - umul_wide (s, s);
    r = __umulhi ((uint32_t)(rem >> 32) + 1, r);
    s = s + r;
    // Denormalize result.
    s = s >> (scal >> 1);
    // Make sure we get the floor correct; can be off by one to either side.
    rem = arg - umul_wide (s, s);
    if ((int64_t)rem < 0) s--;
    else if (rem >= ((uint64_t)s * 2 + 1)) s++;
    return (arg == 0) ? 0 : s;
}


/* ------------------------------------------------------------------------------------------------------------------
 * My code starts here. Don't touch getKGonal or checkPair, but change findc and mess with types as needed (long long
 * vs. uint64_t in particular). Todo: fix the overflow issues.
 * --------------------------------------------------------------------------------------------------------------- */
// General formula for the k-gonal numbers. Do not touch the logic (unless I did something REALLY dumb), but change the type as needed.
__device__ long long int getKGonal(long long a) { return (k % 2 == 1) ? a*((k-2)*a+(4-k))/2 : a*((k/2-1)*a-((k/2-1)-1))/2; }

// Check if a is a perfect square. Potential issues here.
__device__ bool isPerfectSquare(uint64_t a) { 
    uint64_t s = isqrtll(a);
    // Check whether the square of its integer square root is itself.
    return s*s == a; 
}

// Each block takes an a-value, then the threads within that block split up b.
__global__ void checkPair(pr *out) {
    int aidx = blockIdx.x+1;
    int bidx = threadIdx.x;
    for(int a = aidx; a <= abound; a += boxsize) {
        if(a==0) continue;

        // This is not very efficient! You lose a couple of threads per block (depending on parameters and search bounds).
        for(int b = a+bidx; b <= bbound; b += threadsize) {
            long long int aprime = getKGonal(a);
            long long int bprime = getKGonal(b);
            // Write the pair to the output array if it satisfies D(n).
            if(isPerfectSquare(aprime * bprime + n)) out[(a-1)*bbound + b] = thrust::pair(a,b);
        }
    }
}

// We have a and b already, so check c-values.
// Each block takes a pair (a, b) in in, and each thread takes a subset of the search space for c.
__global__ void findc(thrust::device_vector<pr> in, long long int *out) {
    for(int q = blockIdx.x; q <= in.size(); q += boxsize) {
        // Grab the pair - for some reason you can't oneline this.
        pr p = in[q];
        int a = p.first;
        int b = p.second;

        // The "if(x==0) continue;" statements are here to avoid issues with values equalling 0.
        // I don't know whether I still need them but I worry the program will fall apart if I remove them.
        if(a==0 || b==0) continue;
        long long int aprime = getKGonal(a);
        long long int bprime = getKGonal(b);
        for(long long int c = b+threadIdx.x; c <= cbound; c += threadsize) {
            if(c==0) continue;

            // The overflow issues probably kick in around here. This is where you should test for the overflow issues.
            long long int cprime = getKGonal(c);
            if(isPerfectSquare(aprime * cprime + n) && isPerfectSquare(cprime * bprime + n)) out[q] = c;
        }
    }
}

// Predicate for thrust::stable_partition to remove all pairs with zeros below.
struct is_nonzero {
    __host__ __device__
    bool operator()(const pr x) { return x.first != 0 && x.second != 0; }
};


// MAIN -------------------------------------------------------------------------------------------------------------
int main()
{
    // I wish there was an easier way to do this... Standard memory allocation.
    // I don't actually know if I need to instantiate an array on the host side.
    // Check that out if you try to fix the memory overflow issues.
    pr *out, *d_out;
    out = (pr *)malloc(abound*bbound*sizeof(pr));
    hipMalloc((void **)&d_out, abound*bbound*sizeof(pr));
    hipMemcpy(d_out, out, abound*bbound*sizeof(pr), hipMemcpyHostToDevice);

    // Run the first part of the program.
    checkPair<<<boxsize, threadsize>>>(d_out);

    // Absolutely need to wait for that to finish running before continuing.
    hipDeviceSynchronize();


    // THE CODE IS DEFINITELY GOOD UP TO HERE. Past here, uh, no promises.


    // Grab the pointer from the output from the last part and use it to create a thrust::device_vector with the values.
    thrust::device_ptr<pr> d_ptr(d_out);
    thrust::device_vector<pr> d_vec(d_ptr, d_ptr + abound*bbound);
    // Sorts the vector putting all nonzero pairs first and nonzero pairs last, and returns an iterator to the first zero.
    thrust::device_vector<pr>::iterator new_end = thrust::stable_partition(d_vec.begin(), d_vec.end(), is_nonzero());
    // Create a new thrust::device_vector with exactly the nonzero pairs.
    thrust::device_vector<pr> d_vec2(d_vec.begin(), new_end);


    // THE CODE IS VERY NEARLY DEFINITELY GOOD UP TO HERE.


    // Second verse, same as the first. Standard memory allocation, but do I need to allocate memory on the host?
    long long int *out2, *d_out2;
    out2 = (long long int *)malloc(d_vec2.size()*sizeof(long long int));
    hipMalloc((void **)&d_out2, d_vec2.size()*sizeof(long long int));
    hipMemcpy(d_out2, out2, d_vec2.size()*sizeof(long long int), hipMemcpyHostToDevice);

    // Run the second part of the program.
    findc<<<boxsize, threadsize>>>(d_vec2, d_out2);

    // Absolutely need to wait for that to finish running before continuing.
    hipDeviceSynchronize();

    // Copy the output back to the host since we don't need to fanagle with intermediate processing again.
    hipMemcpy(out2, d_out2, d_vec2.size()*sizeof(long long int), hipMemcpyDeviceToHost);

    // Print final result to stdout.
    for(int i = 0; i < d_vec2.size(); i++) {
        // I'm PRETTY sure this line combats the overflow (the less-than in particular) but I'm not 100% sure.
        if(out2[i] <= 0) continue;
        pr p = d_vec2[i];
        cout << p.first << " " << p.second << " " << out2[i] << std::endl;
    }

    return 0;
}